#include "hip/hip_runtime.h"
/*
Copyright 2018 XIAOLIN WANG 

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "NeuralSentSeg.h"
#include "EvaluateSegment.h"
#include "stdlib.h"
#include <list>
#include "Segmentor.h"
using std::list;

namespace sentSeg
{
extern Vocabulary vocab;

void NeuralSentSeg::work()
{
	if(params.mode=="train")
	{
		workTrain();
	}
	else if(params.mode=="apply")
	{
		workApply();
	}
	else if(params.mode=="tune")
	{
		workTune();
	}
	else
	{
		fprintf(stderr, "unknown params.mode %s", params.mode.c_str());
		assert(false);
	}

}

void NeuralSentSeg::workTrain()
{
	XLLibTime startTime=XLLib::startTime();

	vocab.load(params.vocabFile, params.vocabSize);
	if(params.vocabSize!=vocab.size())
	{
		fprintf(stderr, "change vocobSize %d => %d\n", params.vocabSize, vocab.size());
		params.vocabSize=vocab.size();
	}
	batch.init();
	network.init(batch);

	CorpusNss corpus;
	corpus.init(params.trainCorpus, &vocab, params.batchSize);

	CorpusNss  devCorpora;
	devCorpora.init(params.devCorpus, &vocab, params.batchSize);

	int epoch=0;
	Precision trainScore=0;
	int trainNum=0;
	cytonLib::testMode=false;
	Precision bestDevScore=-1000;
	string bestModel;
	corpus.shuffleReset();
	bool corpusTraining=false;
	int numFails=0;

	if(params.loadModel!="")
	{
		bestModel=params.loadModel;
		this->loadModel(bestModel);
		XLLib::printf("loadModel %s", bestModel.c_str());
		Precision tScore=testCorpus(devCorpora);
		bestDevScore=std::max(tScore, bestDevScore);
	}
	if(params.saveModel!="")
	{
		saveModelSetting(params.saveModel);
	}
	int iTest=0;
	bool decay=false;
	XLLib::printf("#Epoch Update Time LearningRate, likelihood Train Dev\n");
	for(int iBatch=0; ; iBatch++)
	{
		assert(testMode==false);

		bool epochEnd;
		bool tRead=corpus.read(batch, corpusTraining);
		if(!tRead)
		{
			epochEnd=true;
			epoch +=1;
			corpus.shuffleReset();
			tRead=corpus.read(batch, corpusTraining);
		}
		else
		{
			epochEnd=false;
		}

		Precision tScore=network.learn();
		trainScore +=tScore;
		trainNum+=batch.getNumTarget();

		if(epochEnd)
		{
			XLLib::printf("#%d %d %s %.2e, likelihood %.3e", epoch, iBatch, XLLib::endTime(startTime).c_str(),
					 params.learningRate, trainScore/trainNum);
			iTest +=1;
			bool success=false;
			string tModel;
			Precision tScore=testCorpus(devCorpora);
			tModel=XLLib::stringFormat("%s/model_e%d_s%d_%.3f", params.saveModel.c_str(), epoch, iBatch, tScore);
			success=(tScore >= bestDevScore+1e-3);
			bool successRelax=(tScore >= bestDevScore+1e-3);
			bestDevScore=std::max(tScore, bestDevScore);

			if(successRelax)
			{
				weightFactory.save(tModel);
				numFails=0;
				XLLib::fileLink(XLLib::fileName(tModel), XLLib::stringFormat("%s/model", params.saveModel.c_str()));
				bestModel=tModel;
				XLLib::printf(" s");
			}

			if(!success)
			{
				decay=true;
				numFails+=1;
				XLLib::printf(" f%d", numFails);
				if(numFails>=3)
				{
					printf("\n");
					break;
				}
				if(!successRelax)
				{
					weightFactory.load(bestModel);
					printf(" load %s\n", XLLib::fileName(bestModel).c_str());
				}
			}

			if(decay)
			{
				params.learningRate *=params.learnRateDecay;
			}

			printf("\n");
			trainScore=0;
			trainNum=0;
		}
	}

	printf("\nbestModel %s\n", bestModel.c_str());

}

void NeuralSentSeg::workApply()
{
	cytonLib::testMode=true;
	cytonLib::batchSize=1;
	params.batchSize=1;
	vector<double>& thresholds=params.thresholds;
	int nf=params.numFutureWords;
	int nf0=1;
	assert(thresholds.size()==nf-nf0+1);

	loadModel(params.loadModel);

	bool gbTestMode=true;
	std::swap(gbTestMode, cytonLib::testMode);
	EvaluateSegment evalSeg;
	XLLibTime startTime=XLLib::startTime();

	Segmentor segmentor;
	segmentor.init(&network, &vocab, &thresholds);

	const string& inputFile=params.input;
	const string& outputFile=params.output;

	FILE* f;
	if(inputFile!="stdin")
	{
		f=fopen(inputFile.c_str(),"r");
	}
	else
	{
		f=stdin;
	}
	if (f == NULL)
	{
		fprintf(stderr, "cannot open the input file %s", inputFile.c_str());
		exit(1);
	}

	FILE* g;
	if(outputFile!="stdout")
	{
		g=fopen(outputFile.c_str(),"w");
	}
	else
	{
		g=stdout;
	}

	char * line = NULL;
	size_t len = 0;
	vector<string> words;
	vector<string> segment;
	while(true)
	{
		ssize_t read = getline(&line, &len, f);
		int tLen=strlen(line);
		while(tLen>0 && line[tLen-1]=='\n' )
		{
			line[tLen-1]=0;
			tLen-=1;
		}

		bool isEnd=false;
		if(read!=-1)
		{
			XLLib::str2list(line, words);
			segmentor.putWords(words);
			isEnd=words.empty();
		}
		else
		{
			isEnd=true;
		}

		while(segmentor.getSegment(isEnd, segment))
		{
			for(unsigned int i=0; i<segment.size(); i++)
			{
				if(i!=0)
				{
					fprintf(g, " ");
				}
				fprintf(g, "%s", segment.at(i).c_str());
			}
			fprintf(g,"\n");
		}
		if(isEnd)
		{
			fprintf(g, "\n");
		}

		if(read==-1)
		{
			break;
		}
	}
	fclose(f);
	fclose(g);
	std::swap(gbTestMode, cytonLib::testMode);
}


void NeuralSentSeg::workTune()
{
	XLLibTime startTime=XLLib::startTime();
	cytonLib::testMode=true;
	cytonLib::batchSize=1;
	params.batchSize=1;

	vector<double>& thresholds=params.thresholds;
	int nf=params.numFutureWords;
	int nf0=1;
	assert(thresholds.size()==nf-nf0+1);

	loadModel(params.loadModel);

	CorpusSi corpora;
	corpora.init(params.devCorpus);

	int nTh=thresholds.size();
	double dTh=0.1;
	double minTh=0.2;
	std::multimap<double, vector<double>> que;
	que.insert(std::pair<double, vector<double>>(0.0, thresholds));
	std::set<vector<double>> dict;
	vector<double> scores;
	vector<string> details;
	double bestScore=-1000;
	string bestLog;
	int tuneSteps=params.tuneSteps;
	int testPeriod=100;
	for(int is=0; ; is++)
	{
		bool searchEnd= (tuneSteps>0 && is>=tuneSteps);

		vector<double> ths;
		if(!searchEnd)
		{
			searchEnd=true;
			while(!que.empty())
			{
				std::multimap<double,vector<double>>::iterator it=que.begin();
				ths.assign(it->second.begin(), it->second.end());
				que.erase(it);

				if(dict.find(ths)==dict.end())
				{
					searchEnd=false;
					break;
				}
			}
		}

		if( (is!=0 && is%testPeriod==0) || searchEnd)
		{
			printf("\n# %2d %d %d\n bestScore %.4f bestLog %s\n", is,
					que.size(), dict.size(),
					bestScore, bestLog.c_str());
			string tFile=XLLib::stringFormat("%s/resTune.%d", params.output.c_str(), is);
			printf("\nwriting %s\n", tFile.c_str());
			vector<string> tDetails(details);
			XLLib::sort(scores, tDetails);
			XLLib::dirPrepare4file(tFile);
			XLLib::writeFile(tFile, tDetails);
		}
		if(searchEnd)
		{
			break;
		}

		double tScore=tuneApply(corpora, ths,  bestScore, bestLog, scores, details);
		assert(scores.size()==details.size());
		dict.insert(ths);

		if(tScore>=bestScore-params.scoreTolerance)
		{
			XLLib::printf(" *");
			for(int k=nTh-1; k>=0; k--)
			{
				for(int k1=0; k1<2; k1++)
				{
					double& th=ths.at(k);
					double dTh1= (k1==0?-1:1)*dTh;
					double th1=th+dTh1;
					if(th1>=minTh && th1<=1.0)
					{
						th+=dTh1;
						bool valid=true;
						for(int it=0; it<nTh-1; it++)
						{
							if(ths.at(it)<ths.at(it+1))
							{
								valid=false;
								break;
							}
						}

						if(valid && dict.find(ths)==dict.end())
						{
							que.insert(std::pair<double, vector<double> >(-tScore, ths));
						}
						th-=dTh1;
					}
				}
			}
		}
		printf("\n");


	}
	printf("\nEnd bestScore %.4f %s\n", bestScore, bestLog.c_str());
}

double NeuralSentSeg::tuneApply(CorpusSi& corpus, vector<double>& thresholds,
		double& bestScore, string& bestLog, vector<double>& scores, vector<string>& details)
{
	int nf=params.numFutureWords;
	int nf0=1;
	assert(thresholds.size()==nf-nf0+1);

	bool gbTestMode=true;
	std::swap(gbTestMode, cytonLib::testMode);
	EvaluateSegment evalSeg;
	vector<string> res;
	ostringstream os;

	XLLib::printf(true, &os, "%s", XLLib::toString_vec(thresholds, "%.2f",":").c_str());
	XLLib::printf(true, &os, " ");

	XLLibTime startTime=XLLib::startTime();
	ofstream* ptrG=NULL;
	evalSeg.reset();
	for(int is=0; is<corpus.samples.size(); is++)
	{
		if(is%10==0)XLLib::printf(".");
		SampleSi& s=corpus.samples.at(is);
		network.apply(s, thresholds, ptrG);
		evalSeg.apply(s, ptrG);
	}
	vector<double> tScores;
	double score=evalSeg.getResult(&os, &tScores);
	scores.push_back(score);
	details.push_back(os.str());
	if(score>bestScore)
	{
		bestScore=score;
		bestLog=os.str();
	}
	std::swap(gbTestMode, cytonLib::testMode);
	return score;
}

void updateDict(std::map<string,int>& dict, const string& key)
{
	std::map<string, int>::iterator it=dict.find(key);
	if(it==dict.end())
	{
		dict[key]=1;
	}
	else
	{
		it->second +=1;
	}
}

void printDict(std::map<string,int>& dict)
{
	vector<string> keys;
	vector<int> vals;
	for(std::map<string, int>::iterator it=dict.begin(); it!=dict.end(); it++)
	{
		keys.push_back(it->first);
		vals.push_back(it->second);
	}

	vector<int> idxs;
	XLLib::sortIndex(vals, idxs);
	for(int k=0; k<idxs.size(); k++)
	{
		int idx=idxs.at(k);
		printf("%s %d\n", keys.at(idx).c_str(), vals.at(idx));
	}
}

Precision NeuralSentSeg::testCorpus(CorpusNss& corpus)
{
	bool gbTestMode=true;
	std::swap(gbTestMode, cytonLib::testMode);
	corpus.kIndex=0;
	int ns=0;
	Precision score=0;
	while((corpus.read(batch, false)))
	{
		score += network.getScore();
		ns+=batch.getNumTarget();
	}
	score /=ns;

	XLLib::printf(" %.3e", score);
	std::swap(gbTestMode, cytonLib::testMode);
	return score;
}


void NeuralSentSeg::saveModelSetting(const string& modelDir)
{
	string tVocab=modelDir+"/vocab";
	XLLib::dirPrepare4file(tVocab);
	vocab.save(tVocab);
	params.saveModelSetting(modelDir);
}

void NeuralSentSeg::loadModel(const string& modelName)
{
	if(params.loadModel=="")
	{
		fprintf(stderr, "loadModel is empty %s\n", params.loadModel.c_str());
		exit(1);
	}
	else
	{
		int i=params.loadModel.rfind('/');
		string modelDir=params.loadModel.substr(0,i);

		string tFile=modelDir+"/vocab";
		vocab.load(tFile, 0);
		if(params.vocabSize!=vocab.size())
		{
			params.vocabSize=vocab.size();
		}
		batch.init();
		network.init(batch);
		weightFactory.load(params.loadModel);
		fprintf(stderr, "loadWeights %s\n", modelName.c_str());
	}

}

} /* namespace sentSeg */
