#include "hip/hip_runtime.h"
/*
Copyright 2018 XIAOLIN WANG 

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "NetworkNss.h"
#include "WeightFactory.h"
#include "ParamsNss.h"

namespace sentSeg
{

extern Vocabulary vocab;



void NetworkNss::init(BatchNss& batch_)
{
	batch=&batch_;
	hx=&batch_.x;
	hy=&batch_.y;

	int nf=params.numFutureWords;
	int nf0=1;
	assert(nf>=nf0);
	targetDim=nf-nf0+2;

	x.copyFrom(*hx);
	Variable* tx=embedding.init("embedding", &x, hx, params.vocabSize, params.hiddenSize);

	tx = lstm.init("lstm", tx, false, params.hiddenSize, params.numLayers, params.dropout);
	lstm.hy.grad.setZero();
	lstm.cy.grad.setZero();

	tx = reshape.init("", tx, tx->length()/params.hiddenSize, params.hiddenSize, 1, 1 );

	tx = dropOut.init("dropout", tx, params.dropout);

	tx=linear.init("linear", tx, targetDim, true);

	tx = softmax.init("softmax", tx);

	assert(tx->n == batch_.y.length());

	weightFactory.init("SGD");
	weightFactory.alloc(5.0);
}


void NetworkNss::forward()
{
	x.copyFrom(*hx);
	embedding.forward();

	lstm.forward();
	{
		Variable*tx =&lstm.y;
		reshape.forward(tx->length()/params.hiddenSize, params.hiddenSize, 1, 1);
	}

	dropOut.forward();

	linear.forward();

	softmax.forward();

	predY.copyFrom(softmax.y);

}

Precision NetworkNss::setTarget()
{
	assert(predY.ni==targetDim);
	int& numTargets=batch->numTargets;
	grad.resize(predY.ni, predY.nj);
	Precision score=0;
	Precision scale=1.0/params.batchSize;

	HostMatInt& targets=*hy;
	//set grad
	assert(grad.nj == targets.length());
	numTargets=0;
	for(int j=0; j<targets.length(); j++)
	{
		int target=targets.at(j);
		for(int i=0; i<predY.ni; i++)
		{
			Precision tPred=predY.at(i,j);
			Precision& tGrad=grad.at(i,j);
			if(target>=0 )
			{
				assert(target<predY.ni);
				if(target==i)
				{
					tGrad=1-tPred;
					score += log(tPred+1e-9);
					numTargets+=1;
				}
				else
				{
					tGrad=-tPred;
				}
				tGrad *=scale;
			}
			else
			{
				tGrad=0;
			}
		}
	}
	assert(softmax.x->length()==grad.length());
	softmax.x->grad.copyFrom(grad.data, grad.length());
	return score;
}

Precision NetworkNss::backward4learn()
{
	Precision score=setTarget();

	linear.backward();

	dropOut.backward();

	lstm.backward();

	embedding.backward();

	return score;
}

void NetworkNss::calculateGradient()
{
	embedding.calculateGradient();

	lstm.calculateGradient();

	linear.calculateGradient();
}


Precision NetworkNss::learn()
{
	assert(cytonLib::testMode==false);

	forward();

	Precision score=backward4learn();

	weightFactory.whole.grad.setZero();
	calculateGradient();

	weightFactory.update(params.learningRate);

	return score;

}

Precision NetworkNss::getScore()
{
	forward();
	Precision score=setTarget();
	return score;
}


void NetworkNss::apply(int* wids, int ni_, int nj_)
{
	hx->copyFrom(wids, ni_, nj_);
	this->forward();
}

void NetworkNss::setContext(bool reset)
{
	if(reset)
	{
		lstm.hx.setZero();
		lstm.cx.setZero();
	}
	else
	{
		lstm.hx.copyFrom(lstm.hy);
		lstm.cx.copyFrom(lstm.cy);
	}

}

void NetworkNss::apply(SampleSi& s, vector<double>& thresholds, ostream* os)
{
	int nf=params.numFutureWords;
	int nf0=1;
	assert(thresholds.size()==nf-nf0+1);

	int maxLen=params.maxSentLen;
	int start=0;
	vector<int>& input=s.srcWids;
	vector<int>& boundaries=s.boundaries;
	vector<bool>& segByMax=s.segByMax;
	double latency=0;
	boundaries.clear();
	segByMax.clear();
	HostMatPrec& probs=predY;
	while(start<input.size())
	{
		int len=std::min(maxLen, (int)input.size()-start);
		int segLen=-1;
		double tLatency=0;
		bool tSegByMax=false;

		{
			this->apply(&input[start], 1, len);
			assert(probs.ni==nf-nf0+2 && probs.nj==len);
			for(int j=nf0; j<len; j++)
			{
				for(int i=probs.ni-1; i>=1; i--)
				{
					Precision tProb=probs.at(i, j);
					if(tProb>=thresholds.at(i-1))
					{
						int tSegLen=j+2-i-nf0;
						if(tSegLen>0)
						{
							segLen=tSegLen;
							tLatency=(segLen-1)*segLen/2+(i-1+nf0)*segLen;
							break;
						}
					}
				}
				if(segLen>0)
				{
					break;
				}
			}

			if(segLen<0)
			{
				if(len>=maxLen)
				{
					Precision maxY=-1;
					int maxJ=-1;
					int maxI=-1;
					for(int j=nf0; j<len; j++)
					{
						for(int i=1; i<probs.ni; i++)
						{
							Precision ty=probs.at(i, j)-thresholds.at(i-1);
							if(ty>maxY)
							{
								int tSegLen=j+2-i-nf0;
								if(tSegLen>=0)
								{
									maxY=ty;
									maxJ=j;
									maxI=i;
								}
							}
						}
					}
					segLen=maxJ+2-maxI-nf0;
					tLatency=(segLen-1)*segLen/2+(len-segLen)*segLen;
					tSegByMax=true;
				}
				else
				{
					segLen=len;
					tLatency= (segLen-1)*segLen/2;
				}
			}
		}
		start+=segLen;
		boundaries.push_back(start);
		segByMax.push_back(tSegByMax);
		latency+=tLatency;
	}

	latency /=input.size();

	s.latency=latency;
}

} /* namespace sentSeg */
