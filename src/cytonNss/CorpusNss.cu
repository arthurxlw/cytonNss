#include "hip/hip_runtime.h"
/*
Copyright 2018 XIAOLIN WANG 

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "CorpusNss.h"
#include "ParamsNss.h"

namespace sentSeg
{

void CorpusNss::init(const string& corpusFile, Vocabulary* vocab_, int batchSize)
{
	vocab=vocab_;

	ifstream f;
	XLLib::fileReader(corpusFile, f);
	string line;
	vector<string> words;
	while (getline(f, line, '\n'))
	{
		XLLib::str2list(line, words);
		if(words.size()<params.maxSentLen && words.size()>0)
		{
			vector<int> sent;
			vocab->getIds(words, sent);
			sents.push_back(sent);
		}
	}
	f.close();

	indexes.clear();
	for(int i=0; i<sents.size(); i++)
	{
		indexes.push_back(i);
	}
	kIndex=0;

	fprintf(stderr, "corpus %d sents %d batches .\n", sents.size(), sents.size()/batchSize);
}

void CorpusNss::reset()
{
	kIndex=0;
}

void CorpusNss::shuffleReset()
{
	std::random_shuffle(indexes.begin(), indexes.end());
	reset();
}

bool CorpusNss::read(BatchNss& batch, bool training)
{
	int nf=params.numFutureWords;

	if(kIndex>=indexes.size())
	{
		return false;
	}
	int batchSize=batch.x.ni;
	int ns=sents.size()-1;
	assert(ns<RAND_MAX);
	vector<int> idxs;
	int maxSentLen=0;
	int kStart=kIndex;
	for(int i=0; i<batchSize; i++)
	{
		if(kIndex>=indexes.size())
		{
			kIndex=kStart;
		}
		int is=indexes.at(kIndex++);
		idxs.push_back(is);
		int len=sents.at(is).size();
		maxSentLen=std::max(len, maxSentLen);
	}

	int len=maxSentLen+nf;

	HostMatInt& x=batch.x;
	HostMatInt& y=batch.y;
	x.resize(batchSize, len);
	y.resize(batchSize, len);

	batch.lens.clear();
	for(int i=0; i<batchSize; i++)
	{
		int is=idxs.at(i);
		vector<int>& sent=sents.at(is);

		int ix=0;
		for(int k=0; k<sent.size(); k++)
		{
			x.at(i, ix) = sent.at(k);
			ix+=1;
		}

		int ixEnd=ix+nf;
		for(int ks=1; ;ks++)
		{
			int is1 = (is+ks) %sents.size();
			if(training)
			{
				is1=rand()%sents.size();
			}
			vector<int>& sent1=sents.at(is1);
			for(int kw=0; kw<sent1.size(); kw++)
			{
				x.at(i, ix) = sent1.at(kw);
				ix+=1;
				if(ix>=ixEnd)
				{
					break;
				}
			}
			if(ix>=ixEnd)
			{
				break;
			}
		}
		assert(ix==ixEnd);

		for(; ix<len; ix++)
		{
			x.at(i, ix)=vocab->empty;
		}

		{
			int nf0=1;
			int tLen=sent.size();
			int tStart=tLen-1+nf0;
			int tEnd=tStart+nf-nf0+1;
			batch.lens.push_back(tLen);
			for(int iy=0; iy<len; iy++)
			{
				int tv=-1;
				if(iy<nf0)
				{
					tv=-1;
				}
				else if(iy<tStart)
				{
					tv=0;
				}
				else if(iy>=tStart && iy<tEnd)
				{
					tv=iy-tStart+1;
				}
				y.at(i, iy)=tv;
			}
		}

	}
	return true;
}



} /* namespace sentSeg */
