#include "hip/hip_runtime.h"
/*
Copyright 2018 XIAOLIN WANG 

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "Segmentor.h"

namespace sentSeg {

void Segmentor::init(NetworkNss* network_, Vocabulary* vocab_, vector<double>* thresholds_)
{
	network=network_;
	vocab=vocab_;
	thresholds=thresholds_;
}


void Segmentor::putWords(const vector<string>& words)
{
	//update nodes;
	int startPos=nodes.size();

	for(vector<string>::const_iterator it=words.begin(); it!=words.end(); it++)
	{
		WordNode node(*it);
		node.wid=vocab->getId(node.word);
		nodes.push_back(node);
	}

	updateScores(startPos);

}

void Segmentor::updateScores(int start)
{
	int len=std::min(params.maxSentLen-start, (int)nodes.size()-start);
	if(len>0)
	{
		if(start==0)
		{
			network->setContext(true);
		}
		else
		{
			network->setContext(false);
		}

		vector<int> input;
		for(int i=start; i<start+len; i++)
		{
			input.push_back(nodes.at(i).wid);
		}
		network->apply(&input[0], 1, len);
		HostMatPrec& probs=network->predY;

		for(unsigned int i=start;i<start+len;i+=1)
		{
			vector<double>& score=nodes.at(i).score;
			score.clear();
			for(int k=0; k<probs.ni; k++)
			{
				double tProb=probs.at(k, i-start);
				score.push_back(tProb);
			}
		}
	}
}

int Segmentor::findBoundary(bool final)
{
	int boundary=-1;
	int nf0=1;
	int maxLen=params.maxSentLen;
	int len=std::min((int)nodes.size(), maxLen);

	if(boundary<0 && len>0 )
	{
		int segLen=-1;
		for(int j=nf0; j<len; j++)
		{
			WordNode& n=nodes.at(j);
			for(int i=n.score.size()-1; i>=1; i--)
			{
				Precision tProb=n.score.at(i);
				if(tProb>=thresholds->at(i-1))
				{
					int tSegLen=j+2-i-nf0;
					if(tSegLen>0)
					{
						segLen=tSegLen;
						break;
					}
				}
			}
			if(segLen>0)
			{
				break;
			}
		}

		if(segLen<0)
		{
			if(len>=maxLen)
			{
				Precision maxY=-1;
				int maxJ=-1;
				int maxI=-1;
				for(int j=nf0; j<len; j++)
				{
					vector<double>& score=nodes.at(j).score;
					for(int i=1; i<score.size(); i++)
					{
						Precision ty=score.at(i)-thresholds->at(i-1);
						if(ty>maxY)
						{
							int tSegLen=j+2-i-nf0;
							if(tSegLen>0)
							{
								maxY=ty;
								maxJ=j;
								maxI=i;
							}
						}
					}
				}
				segLen=maxJ+2-maxI-nf0;
			}
		}

		if(segLen>0)
		{
			boundary=segLen-1;
		}
	}

	if(boundary<0 && !nodes.empty() && final)
	{
		boundary=nodes.size()-1;
	}

	return boundary;

}

bool Segmentor::getSegment(bool final, vector<string>& words)
{
	int position=findBoundary(final);

	bool found=false;
	if(position>=0)
	{
		found=true;

		words.clear();
		int endPos=position+1; //break after this word;
		for(int i=0;i<endPos;i++)
		{
			words.push_back(nodes.at(i).word);
		}
		nodes.erase(nodes.begin(),nodes.begin()+endPos);

		this->updateScores(0);
	}
	return found;
}



} /* namespace simSeg */
