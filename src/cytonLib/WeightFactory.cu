#include "hip/hip_runtime.h"
/*
Copyright 2018 XIAOLIN WANG 

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "WeightFactory.h"
#include "Global.h"
#include "cublasWrapper.h"
#include "utils.h"

namespace cytonLib {

WeightFactory weightFactory;

void WeightFactory::init(const string& method)
{
	if(method=="adam")
	{
		optAdam=true;
		adamGamma=0.9;
		adamGamma2=0.999999;
		adamEpsilon=1e-9;
	}
	else if(method=="SGD")
	{
		optSgd=true;
	}
	else
	{
		assert(false);
	}

}

void WeightFactory::create(Weight& weight, string tag, int ni, int nj)
{
	weight.create(tag, ni, nj);
	weights.push_back(&weight);
}

void WeightFactory::alloc(Precision clipGradient)
{
	int length=0;
	for(int i=0;i<weights.size();i++)
	{
		Weight& w=*weights.at(i);
		fprintf(stderr, "weight%d %s %d*%d\n", i, w.tag.c_str(), w.ni, w.nj);
		length+=w.length();
	}
	whole.resize(length, 1);
	whole.clipGrad=clipGradient;
	fprintf(stderr, "totalWeight %d\n",length);

	int offset=0;
	for(vector<Weight*>::iterator iw=weights.begin();iw!=weights.end();iw++)
	{
		Weight& w=*(*iw);
		w.set(w.ni, w.ni, w.nj, whole.data+offset, whole.grad.data+offset);
		offset+=w.length();
	}


	whole.initRandom(-global.initFactor, global.initFactor);
	if(optAdam)
	{
		momentum.resize(whole.ni, whole.nj);
		momentum.setZero();
		gradientVariance.resize(whole.ni, whole.nj);
		gradientVariance.setZero();
		dWeight.resize(whole.ni, whole.nj);
	}
	else if(optSgd)
	{
	}
	else
	{
		assert(false);
	}

}

void WeightFactory::clearGrad()
{
	whole.grad.setZero();
}


__global__
void weightFactory_update_adam(Precision* grad, Precision* gradMomentum, Precision* gradVar,  Precision* weight, Precision* dWeight, int len,
		Precision gamma,Precision gamma2, Precision epsilon, Precision lambda )
{
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	if(i<len)
	{
		Precision& g=grad[i];
		Precision& gm=gradMomentum[i];
		Precision& gv=gradVar[i];
		Precision& w=weight[i];
		Precision& dw=dWeight[i];

		gm=(1-gamma)*g+gamma*gm;
		gv=(1-gamma2)*g*g+gamma2*gv;

		dw= gm/(sqrt(gv)+epsilon)*lambda;
		w += dw;
	}
}


void WeightFactory::update(Precision lambda)
{
	int len=whole.length();
	Precision pnFactor=sqrt(1.0/whole.length());

	if(whole.clipGrad>0)
	{
		whole.grad.clip(whole.clipGrad);
	}
	if(optAdam)
	{
		Precision step=global.batch;
		Precision tf=sqrt(1.0-std::pow(adamGamma2, step)) / (1.0-std::pow(adamGamma, step));
		weightFactory_update_adam<<<ceil(len, blockSize), blockSize>>>(whole.grad.data, momentum.data, gradientVariance.data, whole.data, dWeight.data, len,
				adamGamma, adamGamma2, adamEpsilon, lambda*tf);
	}
	else if(optSgd)
	{
		checkError(cublasXaxpy(global.cublasHandle, whole.length(), &lambda, whole.grad.data, 1, whole.data, 1));
	}
	else
	{
		assert(false);
	}
}

void WeightFactory::save(const string& fileName)
{
	XLLib::dirPrepare4file(fileName);
	std::ofstream f(fileName.c_str());

	f<<"##"<<"WeightFactory"<<"\n";
	whole.save(f);
	f.close();
}

void WeightFactory::load(const string& fileName)
{
	if(!XLLib::fileExists(fileName))
	{
		fprintf(stderr, "Error: model file %s does not exist.\n", fileName.c_str());
		assert(false);
	}

	ifstream f(fileName.c_str());
	string tTag=string("##WeightFactory");
	checkFile(f,tTag);
	whole.load(f);
	f.close();
}

} /* namespace cytonLib */
