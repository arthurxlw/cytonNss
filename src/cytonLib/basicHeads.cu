/*
Copyright 2018 XIAOLIN WANG 

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/



#include "basicHeads.h"
#include "Global.h"


namespace cytonLib
{

hipdnnDataType_t cudnnDataType=HIPDNN_DATA_FLOAT;

// Define some error checking macros.
hipError_t checkError_(hipError_t stat, const char *file, int line)
{
	if (stat != hipSuccess)
	{
		string tErr=hipGetErrorString(stat);
		if(tErr!="driver shutting down")
		{
			fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
			assert(false);
			exit(1);
		}
		else
		{
		}
	}
	return stat;
}

hipdnnStatus_t checkError_(hipdnnStatus_t stat, const char *file, int line)
{
	if (stat != HIPDNN_STATUS_SUCCESS)
	{
		fprintf(stderr, "cuDNN Error: %s %s %d\n", hipdnnGetErrorString(stat), file, line);
		assert(false);
		exit(1);
	}
	return stat;
}

hipblasStatus_t checkError_(hipblasStatus_t stat, const char *file, int line)
{
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "cublas Error: %d %s %d\n", stat, file, line);
		assert(false);
		exit(1);
	}
	return stat;
}

hiprandStatus_t checkError_(hiprandStatus_t stat, const char *file, int line)
{
	if (stat != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "hiprand Error: %s %d\n",  file, line);
		assert(false);
		exit(1);
	}
	return stat;
}


static const char *_cusolverGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
        case HIPSOLVER_STATUS_SUCCESS:
            return "CUSOLVER_SUCCESS";

        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            return "HIPSOLVER_STATUS_NOT_INITIALIZED";

        case HIPSOLVER_STATUS_ALLOC_FAILED:
            return "HIPSOLVER_STATUS_ALLOC_FAILED";

        case HIPSOLVER_STATUS_INVALID_VALUE:
            return "HIPSOLVER_STATUS_INVALID_VALUE";

        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            return "HIPSOLVER_STATUS_ARCH_MISMATCH";

        case HIPSOLVER_STATUS_EXECUTION_FAILED:
            return "HIPSOLVER_STATUS_EXECUTION_FAILED";

        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            return "HIPSOLVER_STATUS_INTERNAL_ERROR";

        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

    }

    return "<unknown>";
}

hipsolverStatus_t checkError_(hipsolverStatus_t stat, const char *file, int line)
{
    if(HIPSOLVER_STATUS_SUCCESS != stat) {
        fprintf(stderr, "cusolver error: %s %d, error %d %s\n", file, line,
        		stat, _cusolverGetErrorEnum(stat));
        assert(0);
    }
    return stat;
}

void checkFile(ifstream& f, const string t)
{
	string line;
	while(getline(f, line))
	{
		if(!line.empty())
		{
			break;
		}
	}
	bool right=line==t;
	assert(right);
}

}
