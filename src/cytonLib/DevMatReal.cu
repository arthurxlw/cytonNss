#include "hip/hip_runtime.h"
/*
Copyright 2018 XIAOLIN WANG 

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "HostMatReal.h"
#include "DevMatReal.h"
#include "cublasWrapper.h"
#include "Global.h"

namespace cytonLib
{


template<typename T>
DevMatReal<T>::DevMatReal()
{
}

template<typename T>
DevMatReal<T>::DevMatReal(size_t ni, size_t nj)
{
	this->resize(ni, nj);
}

template<typename T>
void DeviceMatrix<T>::setZero()
{
	if(this->continuous())
	{
		checkError(hipMemset(this->data, 0, this->length()*sizeof(T)));
	}
	else
	{
		for(size_t j=0; j<this->nj; j++)
		{
			checkError(hipMemset(this->data+this->stride*j, 0, this->ni*sizeof(T)));
		}
	}
}

template<typename T>
void DevMatReal<T>::initRandom()
{
	size_t len=this->length();
    checkError(curandGenerateUniformX(global.curandGenerator, this->data, len));
}

template<typename T>
__global__ void initRandom_kernel(T *data, size_t n, T a, T b) {
    size_t i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < n)
    {
    	T& t=data[i];
    	t= a*t+b;
    }
}

template<typename T>
void DevMatReal<T>::initRandom(T low, T up)
{
	this->initRandom();
	size_t n=this->length();
	T a=(up-low);
	T b=low;
	initRandom_kernel<<<ceil(n, blockSize), blockSize>>>(this->data, n, a, b);
	checkError(hipGetLastError());
}

template<typename T>
void DevMatReal<T>::add(T* mat, T alpha)
{
	checkError(cublasXaxpy(global.cublasHandle, this->length(), &alpha,
			mat, 1, this->data, 1));
}

template<typename T>
void DevMatReal<T>::scale(T a)
{
	assert(this->continuous());
	checkError(cublasXscal(global.cublasHandle, this->length(), &a,
			this->data, 1));
}

template<typename T>
void DevMatReal<T>::update(T* mat, T a, T b)
{
	checkError(cublasXscal(global.cublasHandle, this->length(), &a,
			this->data, 1));

	checkError(cublasXaxpy(global.cublasHandle, this->length(), &b,
			mat, 1, this->data, 1));
}

template<typename T>
void DevMatReal<T>::addTo(T* mat, T alpha)
{
	checkError(cublasXaxpy(global.cublasHandle, this->length(), &alpha,
			this->data, 1, mat, 1));
}


template<typename T>
T DevMatReal<T>::getNorm()
{
	assert(this->continuous());
	T res=0;
	checkError(cublasXnrm2(global.cublasHandle, this->length(), this->data, 1, &res));
	return res;
}

template<typename T>
T DevMatReal<T>::clip(T threshold)
{
	T res=this->getNorm();
	if(res>threshold)
	{
		this->scale(threshold/res);
	}
	return res;
}

template<typename T>
T DevMatReal<T>::max() const
{
	HostMatReal<T> mat;

	T ans;
	if(this->length()>4096)
	{
		thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(this->data);
		thrust::device_ptr<T> iter =
			thrust::max_element(dev_ptr, dev_ptr+this->length());
		ans=*iter;
	}
	else
	{
		mat.copyFrom(*this);
		ans=mat.max();
	}
	return ans;
}

template<typename T>
__global__
void devMatReal_reduceCols_min(
    T *result, const T *mat, const MatrixDim d)
{
  __shared__ T sdata[CU1DBLOCK];
  const size_t tid = threadIdx.x;
  const size_t i = blockIdx.x;
  const size_t row_start = i * d.stride;

  T tdata = sizeof(T) == sizeof(float) ? HIP_INF_F : HIP_INF;
  for (size_t j = tid; j < d.ni; j += CU1DBLOCK) {
    tdata = fmin(tdata, mat[row_start + j]);
  }
  sdata[tid] = tdata;
  __syncthreads();

  // Tree reduce
# pragma unroll
  for (size_t shift = CU1DBLOCK / 2; shift > warpSize; shift >>= 1) {
    if (tid < shift)
      sdata[tid] = fmin(sdata[tid], sdata[tid + shift]);
    __syncthreads();
  }

  // Reduce last warp. Threads implicitly synchronized within a warp.
  if (tid < warpSize) {
    for (size_t shift = warpSize; shift > 0; shift >>= 1)
      sdata[tid] = fmin(sdata[tid], sdata[tid + shift]);
  }

  // Output to vector result.
  if (tid == 0) {
    result[i] = sdata[0];
  }
}

template<typename T>
T DeviceMatrix<T>::min() const
{
	HostMatReal<T> mat;

	if(this->length()>4096)
	{
		DevMatReal<T> col_min(this->nj, 1);
		 devMatReal_reduceCols_min<<<this->nj, CU1DBLOCK>>>(col_min.data, this->data, this->dim());
		T ans = col_min.min();
	}
	else
	{
		mat.copyFrom(*this);
	}
	return mat.min();
}

template class DevMatReal<double>;
template class DevMatReal<float>;


} /* namespace cytonLib */
